#include "hip/hip_runtime.h"
#include "original.h"
#include <chrono>

int main(int argc, char ** argv) {

    ///  The A and B matrix
    int * A;
    int * B; 

    ///  The output C matrix
    int * TiledGPUC;
    int * UntiledGPUC; 
    int * CPUC;

	/// Matrix in GPU
    int * GPUA;
    int * GPUB;
    int * GPUC;

    ///  Allocate Inputs
    A = (int *) malloc(sizeof(int)*N*N);
    B = (int *) malloc(sizeof(int)*N*N);

    ///  Allocate Results
    TiledGPUC = (int *) malloc(sizeof(int)*N*N);
    UntiledGPUC = (int *) malloc(sizeof(int)*N*N);
    CPUC = (int *) malloc(sizeof(int)*N*N);

    srand(time(NULL));
    ///  Matrix Initialization
    #pragma omp parallel for default(none) shared(N,A,B)
    for (int i = 0; i < N*N; i++)
    {
        A[i]=rand() % 5;
        B[i]=rand() % 5;
    }

    ///  Allocating GPU memory
    hipMalloc((void **)&GPUA, sizeof(int)*N*N);
    hipMalloc((void **)&GPUB, sizeof(int)*N*N);
    hipMalloc((void **)&GPUC, sizeof(int)*N*N);

    ///  Copy memory to the GPU
    hipMemcpy(GPUA, A, sizeof(int)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(GPUB, B, sizeof(int)*N*N, hipMemcpyHostToDevice);

    ///  Initialize the grid and block dimensions
    ///  Number of Blocks required
    dim3 Grid((N/Tile) + 1, (N/Tile) + 1, 1);
    /// Number of threads in each block
    dim3 Block(Tile, Tile, 1);

    // Initialize timer
    auto start = std::chrono::high_resolution_clock::now();

    /// Launch the GPU Tiled Kernel
    matMultiplyTiled<<<Grid, Block>>>(N,GPUA, GPUB, GPUC);

    // Finish timer
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> duration_ms = end - start;
    printf("Tiled GPU Test, N: %d duration %f ms\n",N,duration_ms.count());
    fflush(stdout); 


    hipDeviceSynchronize(); 
    ///  Copy the results in GPU memory back to the CPU
    hipMemcpy(TiledGPUC, GPUC, sizeof(int)*N*N, hipMemcpyDeviceToHost);

    //reset GPUC
    hipFree(GPUC);
    hipMalloc((void **)&GPUC, sizeof(int)*N*N);

    // Initialize timer
    start = std::chrono::high_resolution_clock::now();

    /// Launch the GPU UnTiled Kernel
    matMultiplyGPU<<<Grid, Block>>>(N,GPUA, GPUB, GPUC);

    // Finish timer
    end = std::chrono::high_resolution_clock::now();

    duration_ms = end - start;
    printf("Untiled GPU Test, N: %d duration %f ms\n",N,duration_ms.count());
    fflush(stdout); 

    hipDeviceSynchronize(); 
    ///  Copy the results in GPU memory back to the CPU
    hipMemcpy(UntiledGPUC, GPUC, sizeof(int)*N*N, hipMemcpyDeviceToHost);

    // Initialize timer
    start = std::chrono::high_resolution_clock::now();

    /// CPU mat Multiply
    matMultiplyCPUOMP(N, A, B, CPUC);

    // Finish timer
    end = std::chrono::high_resolution_clock::now();

    duration_ms = end - start;
    printf("CPU Test, N: %d duration %f ms\n",N,duration_ms.count());
    fflush(stdout); 

    /// Verify both
    printf("1 ");
    fflush(stdout); 
    printf("Untiled GPU vs CPU %s \n", 
        verifyMatrix(N,UntiledGPUC,CPUC) ? "true" : "false");
    printf("2 ");
    fflush(stdout); 
    printf("Tiled GPU vs CPU %s \n",  
        verifyMatrix(N,TiledGPUC,CPUC) ? "true" : "false");

    ///  Free the GPU memory
    hipFree(GPUA);
    hipFree(GPUB);
    hipFree(GPUC);

    /// Free the Pointer Memory
    free(A);
    free(B);
    free(TiledGPUC);
    free(UntiledGPUC);
    free(CPUC);

    return 0;
}

bool verifyMatrix(int N, int * C1,int * C2){
    for (int i=0; i < N*N; i++) {
        if (C1[i]  != C2[i] ) {
            printf("%d %d %d \n",i,C1[i],C2[i]);
            return false; 
        }
    }
    return true;
}

void matMultiplyCPUOMP(int N,int*a,int*b,int*c){
    int result; 
    #pragma omp parallel for default(none) shared(N,a,b,c) private(result)
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            result= 0;
            for(int n=0;n < N;n++){
                result+=a[i*N+n]*b[n*N+j]; 
            }
            c[i*N+j]=result;
        }
    }
}

__global__
void matMultiplyGPU(int N, int * a, int * b, int * c){
    // We get the index of the current data 
    unsigned int threadx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int thready = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int threadxy = thready * N + threadx;

    // Then we get the col and row
    int row = threadxy / N;
    int col = threadxy % N;

    if(row < N && col < N){

        // Then we multiply and add each one of them
        int result = 0;
        for(int i=0;i<N;i++){
            result +=a[row*N+i]*b[i*N+col];
        }

        c[row*N + col]=result;

    }
}

__global__
void matMultiplyTiled(int N, int * a, int * b, int * c){
    ///  Tile size to store elements in shared memory
    __shared__ int sharedA[Tile][Tile];   
    __shared__ int sharedB[Tile][Tile];

    /// To generate ids of threads.
    int Row = blockDim.y*blockIdx.y + threadIdx.y; 
    int Col = blockDim.x*blockIdx.x + threadIdx.x;

    int cvalue = 0.0;

    sharedA[threadIdx.y][threadIdx.x] = 0.0;
    sharedB[threadIdx.y][threadIdx.x] = 0.0;

    /// copy into shared and then calculate 
    for (int k = 0; k < (((N - 1)/ Tile) + 1); k++){
		/// copy Data to Tile from Matrix (Global Memory to Shared Memory)
		if ( (Row < N) && (threadIdx.x + (k*Tile)) < N) {
			sharedA[threadIdx.y][threadIdx.x] =
				 a[(N*N) + threadIdx.x + (k*Tile)];
        }
        /// due to the matrix not always being a multiple
        else
        {
            sharedA[threadIdx.y][threadIdx.x] = 0.0;
        }

        /// copy Data to Tile from Matrix (Global Memory to Shared Memory)
        if ( Col < N && (threadIdx.y + k*Tile) < N) {
            sharedB[threadIdx.y][threadIdx.x] =
				b[(threadIdx.y + k*Tile)*N + Col];
        }
        /// due to the matrix not always being a multiple
        else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0;
        }

        /// Wait for all partials to be calculated
        __syncthreads();

        /// Multiplying Elements present in tile
        for (int j = 0; j < Tile; ++j)
        {
            cvalue += sharedA[threadIdx.y][j] * sharedB[j][threadIdx.x];
        }
    }

    /// Saving Final result into Matrix c
    if (Row < N && Col < N) {
        c[Row*N + Col] = cvalue;
    }
}
